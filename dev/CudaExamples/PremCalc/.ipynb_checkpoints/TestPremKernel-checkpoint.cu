#include "hip/hip_runtime.h"
#include "DataManager.h"

//extern "C" {
#include <hipblas.h>
//#include "cublas_utils.h"
//}

using namespace std;

#define MAX(a, b)  (((a) > (b)) ? (a) : (b)) 


void getPRem(DataManager &DM, DeviceDataPXgICalc *d_pDataPXgI);


int main()
{
    DataManager DM;
    float * pRem; //pRem probs in computer.
    DeviceDataPXgICalc d_pDataPXgI; //Data to use for calculations.
    
    DM.loadDataFromCSV("/home/jkipen/ProtInfGPU/dev/P_X_giv_I_rel_w_GPU/RepVars/"); //Loads the test data
    //cout << to_string(DM.InputData.NFexpForI[0]) << "This was N first ";
    DM.dataToGPU(&d_pDataPXgI); //Copies input data to GPU and allocates for internal variables 
    
    getPRem(DM, &d_pDataPXgI); //Calculation of Prem
    pRem = (float *) malloc(sizeof(float)*DM.n_reads);
    hipMemcpy(pRem,d_pDataPXgI.d_pRem, sizeof(float)*DM.n_reads, hipMemcpyDeviceToHost );
    for(unsigned int i=0;i<DM.n_reads;i++)
        cout << to_string(pRem[i]) << ", ";
    DM.free(&d_pDataPXgI);
    free(pRem);
    return 0;
}

void getPRem(DataManager &DM, DeviceDataPXgICalc *d_pDataPXgI)
{
    float alpha,beta;
    alpha=-1;beta=1; //Parameters for gemv
    unsigned int m,n,max; //Matrix size. CUDA-BLAS is on column major format so its a bit tricky
    hipblasHandle_t handle;
    hipblasOperation_t trans=HIPBLAS_OP_T; //Transpose to have column-major format of the transposed ( A is n_sparxn_reads)
    hipblasStatus_t status;
    float *d_x;
    float *x;
    
    m=DM.n_sparsity;
    n=DM.n_reads;
    max=MAX(m,n);
    
    
    status=hipblasCreate(&handle);
    //Sets ones vector in device x:
    hipMalloc(&d_x, sizeof(float)*m);
    x= (float *) malloc(sizeof(float)*max);
    for(unsigned int i=0;i<max;i++)
        x[i]=1;
    hipMemcpy(d_x, x, sizeof(float)*m, hipMemcpyHostToDevice);
    hipMemcpy(d_pDataPXgI->d_pRem, x, sizeof(float)*n, hipMemcpyHostToDevice); //ones in beta, so we do 1-sum(ps).
    //gemv: y= (alpha)*op(A)@x+ beta*y; where A is mxn matrix, x and y are vectors nx1
    status = hipblasSgemv( handle, trans,
                                m, n,
                                &alpha,
                                d_pDataPXgI->d_TopNFluExpScores, m, 
                                d_x, 1,
                                &beta,
                                d_pDataPXgI->d_pRem, 1);//lda is number of columns
                                
    hipblasDestroy(handle);
}