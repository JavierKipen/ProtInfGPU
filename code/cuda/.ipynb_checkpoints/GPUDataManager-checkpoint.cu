#include "hip/hip_runtime.h"
#include "GPUDataManager.h"
#include <iostream>
#include <vector>
#include <algorithm>    // std::max


using namespace std;

GPUDataManager::GPUDataManager()
{
}
  
bool GPUDataManager::allocateForNumberOfReads(DatasetMetadata *pDM,DeviceData *pdevData, DeviceData **d_ppdevData)
{
    bool retVal=false;
    unsigned long nReadsMax=pDM->nReadsTotal;//In NReads Total is actually stored the number of reads we are using for computing!
    onesVecLen=std::max({nReadsMax,(unsigned long)pDM->nSparsity,(unsigned long)pDM->nProt}); 
    if(!hipMalloc(&(pdevData->d_NFexpForI), sizeof(unsigned int)*pDM->nProt))
        if(!hipMalloc(&(pdevData->d_TopNFluExpId), (unsigned long)sizeof(unsigned int)*nReadsMax*(unsigned long)pDM->nSparsity))
            if(!hipMalloc(&(pdevData->d_FexpIdForI), sizeof(unsigned int)*pDM->fluExpIdForI.size()))
                if(!hipMalloc(&(pdevData->d_PFexpForI), sizeof(float)*pDM->fluExpIdForI.size()))
                    if(!hipMalloc(&(pdevData->d_TopNFluExpScores), (unsigned long)sizeof(float)*nReadsMax*(unsigned long)pDM->nSparsity))
                        if(!hipMalloc(&(pdevData->d_pRem), sizeof(float)*nReadsMax))
                            if(!hipMalloc(&(pdevData->d_MatAux), (unsigned long)sizeof(float)*nReadsMax*(unsigned long)pDM->nProt))
                                if(!hipMalloc(&(pdevData->d_ones), sizeof(float)*onesVecLen))
                                    if(!hipMalloc(d_ppdevData, sizeof(*pdevData)))
                                        if(!hipMalloc(&(pdevData->d_PIEst), sizeof(float)*pDM->nProt))
                                            if(!hipMalloc(&(pdevData->d_VecAux), sizeof(float)*onesVecLen))
                                                retVal=true;
    return retVal;
}

unsigned long GPUDataManager::maxReadsToCompute(DatasetMetadata *pDM,unsigned long nBytesLim)
{ //Given a byte limit, we can calculate how many reads could be computed at the same update.
    unsigned long metadataBytes = sizeof(unsigned int)* (pDM->fluExpIdForI.size(),pDM->nFluExpForI.size()+5)
                                    + sizeof(float)* pDM->probFluExpForI.size();//+4 unsigned int for the nsparsity,nreads... . 
    unsigned long calcVariables = sizeof(float)* (2*pDM->nProt); //VecAux and PIEst has length nProt. Problem with d_ones: Can depend or not of length of reads (we dont consider it to be simpler, but its ok because its much less size than other variables.
    unsigned long bytesToAllocPerRead= sizeof(float) * (1+(pDM->nProt)+1+(pDM->nSparsity)) + sizeof(unsigned int) *pDM->nSparsity; //Floats corresponding to  d_pRem, *d_MatAux, *d_ones and d_TopNFluExpScores, and uint d_TopNFluExpScores.
    
    return (nBytesLim-calcVariables-metadataBytes)/bytesToAllocPerRead;
}


void GPUDataManager::metadataToGPU(DatasetMetadata *pDM,DeviceData *pdevData, DeviceData *d_pdevData)
{
    hipMemcpy(pdevData->d_NFexpForI, pDM->nFluExpForI.data(), sizeof(unsigned int)*pDM->nProt, hipMemcpyHostToDevice);
    hipMemcpy(pdevData->d_FexpIdForI, pDM->fluExpIdForI.data(), sizeof(unsigned int)*pDM->fluExpIdForI.size(), hipMemcpyHostToDevice);
    hipMemcpy(pdevData->d_PFexpForI, pDM->probFluExpForI.data(), sizeof(float)*pDM->fluExpIdForI.size(), hipMemcpyHostToDevice);
    pdevData->nSparsity=pDM->nSparsity;
    pdevData->nReadsMax=pDM->nReadsTotal;//In NReads Total is actually stored the number of reads we are using for computing!
    pdevData->nReadsProcess=pDM->nReadsTotal; //Assumes that nReadProcess will be max (can change in the processing calls.
    pdevData->nFluExp=pDM->nFluExp;
    pdevData->nProt=pDM->nProt;
    createOnesVec(pdevData);
    unsigned int size_data=sizeof(*pdevData);
    hipMemcpy(d_pdevData, pdevData, size_data, hipMemcpyHostToDevice);
}

void GPUDataManager::loadNewDataToGPU(PNewData pNewData,DeviceData *pdevData,DeviceData *d_pdevData)
{
    pdevData->nReadsProcess=pNewData.nReads;//Amount of reads to process
    hipMemcpy(&(d_pdevData->nReadsProcess), &(pNewData.nReads), sizeof(unsigned int), hipMemcpyHostToDevice); //Copies reads to process to device
    hipMemcpy(pdevData->d_TopNFluExpId, pNewData.pTopNFluExpIds, sizeof(unsigned int)*pdevData->nReadsProcess*pdevData->nSparsity, hipMemcpyHostToDevice);
    hipMemcpy(pdevData->d_TopNFluExpScores, pNewData.pTopNFluExpScores, sizeof(float)*pdevData->nReadsProcess*pdevData->nSparsity, hipMemcpyHostToDevice);
    hipMemcpy(pdevData->d_PIEst, pNewData.pPIEst, sizeof(float)*pdevData->nProt, hipMemcpyHostToDevice); //Could be passed less times!
}


void GPUDataManager::retrieveOutput(float * updateVectorOut,DeviceData *devData)
{
    hipMemcpy(updateVectorOut, devData->d_VecAux, sizeof(float)*devData->nProt, hipMemcpyDeviceToHost); //The update is contained in the auxiliar vector.
}

void GPUDataManager::createOnesVec(DeviceData *pdevData)
{
    vector<float> ones(onesVecLen, 1);
    hipMemcpy(pdevData->d_ones, ones.data(), sizeof(float)*onesVecLen, hipMemcpyHostToDevice);
    hipMemcpy(pdevData->d_VecAux, ones.data(), sizeof(float)*onesVecLen, hipMemcpyHostToDevice); //We also initialize dvecAux
}
void GPUDataManager::freeData(DeviceData *pdevData, DeviceData *d_pdevData)
{
    hipFree( pdevData->d_NFexpForI );
    hipFree( pdevData->d_TopNFluExpId );
    hipFree( pdevData->d_FexpIdForI );
    hipFree( pdevData->d_PFexpForI );
    hipFree( pdevData->d_TopNFluExpScores );
    hipFree( pdevData->d_pRem );
    hipFree( pdevData->d_MatAux );
    hipFree( pdevData->d_ones );
    hipFree( pdevData->d_PIEst );
    hipFree( pdevData->d_VecAux );
    hipFree( d_pdevData );
}

GPUDataManager::~GPUDataManager()
{
}
